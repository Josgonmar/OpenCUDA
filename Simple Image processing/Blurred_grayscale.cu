#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <opencv2/core/core.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>
#include "hip/hip_runtime.h"
#include ""

__constant__ float gaussianKernelDevice[256];

__global__ void convolution(int rows, int cols, int kRows, int kCols, unsigned char* input, unsigned char* output) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;

    int pWidth = kCols / 2;
    int pHeight = kRows / 2;
    float sum = 0.0;

    if (idx >= pWidth && idx < cols-pWidth && idy < rows-pHeight && idy >= pHeight) {
        for (int i = 0; i < kRows; i++) {
            for (int j = 0; j < kCols; j++) {
                sum += gaussianKernelDevice[j + i * kCols] * input[(idx + j - pWidth) + (idy + i - pHeight) * cols];
            }
        }
        output[idx + idy * cols] = (unsigned char)sum;
    }
}

__global__ void rgb2gray(int rows, int cols, int channels, unsigned char* input, unsigned char* output) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;

    if (idx < cols && idy < rows) {
        unsigned char r = input[(idx + idy * cols) * channels];
        unsigned char g = input[(idx + idy * cols) * channels + 1];
        unsigned char b = input[(idx + idy * cols) * channels + 2];
        output[idx + idy * cols] = r * 0.299f + g * 0.587f + b * 0.114f;
    }
}

int main(int argc, char** argv) {
    // Open a webcamera
    cv::VideoCapture camera(0);
    cv::Mat frame;
    if (!camera.isOpened()) return -1;

    // Create the cuda event timers 
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    camera >> frame;
    cv::Mat output(frame.size().height,frame.size().width,CV_8U);
    output = 0; //Initialize the cv::Mat with zeros, so it can be overwrited with cudaMat data at the end

    const float gaussianKernel5x5[25] =
    {
        1.f / 273.f,  4.f / 273.f,  7.f / 273.f,  4.f / 273.f, 1.f / 273.f,
        4.f / 273.f,  16.f / 273.f, 26.f / 273.f,  16.f / 273.f, 4.f / 273.f,
        7.f / 273.f, 26.f / 273.f, 41.f / 273.f, 26.f / 273.f, 7.f / 273.f,
        1.f / 273.f,  4.f / 273.f,  7.f / 273.f,  4.f / 273.f, 1.f / 273.f,
        4.f / 273.f,  16.f / 273.f, 26.f / 273.f,  16.f / 273.f, 4.f / 273.f,
    };
    hipMemcpyToSymbol(HIP_SYMBOL(gaussianKernelDevice), gaussianKernel5x5, sizeof(gaussianKernel5x5), 0);

    unsigned char* cuda_input = NULL;
    unsigned char* cuda_output = NULL;
    hipMalloc(&cuda_input, sizeof(unsigned char) * frame.size().width * frame.size().height * frame.channels());
    hipMalloc(&cuda_output, sizeof(unsigned char) * frame.size().width * frame.size().height);
    // Loop while capturing images
    while (1)
    {
        // Capture the image and store a gray conversion to the gpu
        camera >> frame;
        hipMemcpy(cuda_input, frame.data, sizeof(unsigned char) * frame.size().width * frame.size().height * frame.channels(), hipMemcpyHostToDevice);
        hipMemset(cuda_output, 0, sizeof(unsigned char) * frame.size().height * frame.size().width);
        // Record the time it takes to process
        hipEventRecord(start);
        {
            // convolution kernel launch parameters
            dim3 cblocks(frame.size().width / 16, frame.size().height / 16);
            dim3 cthreads(16, 16);
            rgb2gray << < cblocks, cthreads >> > (frame.size().height, frame.size().width, frame.channels(), cuda_input, cuda_output);
            convolution << < cblocks, cthreads >> > (frame.size().height, frame.size().width, 5, 5, cuda_output, cuda_output);
            hipMemcpy(output.data, cuda_output, sizeof(unsigned char) * frame.size().width * frame.size().height, hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
        }
        hipEventRecord(stop);

        // Display the elapsed time
        float ms = 0.0f;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&ms, start, stop);
        std::cout << "Elapsed GPU time: " << ms << " milliseconds" << std::endl;

        // Show the results
        char line[99];
        sprintf_s(line, 99, "FPS: %f", 1000 / ms);
        cv::putText(output, line, cv::Point(50, 50), cv::FONT_HERSHEY_COMPLEX_SMALL, 1, cv::Scalar(255, 0, 0), 1, false);
        cv::imshow("Original", frame);
        cv::imshow("Result", output);
        // Spin
        if (cv::waitKey(1) == 27) break;
    }

    // Exit
    hipFree(cuda_input);
    hipFree(cuda_output);

    return 0;
}